#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define THREAD_PER_BLOCK 256 // 这里定义每个block中的线程数量，同时每个block中处理数组的大小也是这个

__global__ void reduce0(float *d_input, float *d_out)
{
    // 确定索引起始点
    float *inputPtr = d_input + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // reduce计算
    for (int i = 1; i < blockDim.x; i *= 2)
    {
        if (tid % (i * 2) == 0)
        {
            inputPtr[tid] += inputPtr[tid + i];
        }
        __syncthreads();
    }

    // 保存结果
    if (tid == 0)
    {
        d_out[blockIdx.x] = inputPtr[0];
    }
}

bool check(float *a, float *b, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (fabs(a[i] - b[i]) > 0.005)
        {
            printf("The ans is wrong!\n");
            return false;
        }
    }

    printf("It's all right!\n");
    return true;
}

int main(int argc, char const *argv[])
{
    // 定义常量
    constexpr int N = 32 * 1024 * 1024;
    constexpr int BLOCK_NUM = N / THREAD_PER_BLOCK;

    // 定义变量
    float *input, *d_input;
    float *out, *d_out;
    float *res;

    // 初始化CPU内存
    input = (float *)malloc(N * sizeof(float)); // 输入的数据内存大小
    for (int i = 0; i < N; i++)
    {
        input[i] = 1;
    }
    out = (float *)malloc(BLOCK_NUM * sizeof(float))     // 输出的数据内存大小
        res = (float *)malloc(BLOCK_NUM * sizeof(float)) // 输出的数据内存大小

        // 初始化GPU内存
        hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_out, BLOCK_NUM * sizeof(float));
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    // 调用核心函数
    reduce0<<<dim3(BLOCK_NUM, 1), dim3(THREAD_PER_BLOCK, 1)>>>(d_input, d_out);

    // 复制值到CPU
    hipMemcpy(out, d_out, BLOCK_NUM * sizeof(float), hipMemcpyDeviceToHost);

    // 计算CPU端的值
    for (int i = 0; i < BLOCK_NUM; i++)
    {
        float cur = 0;
        for (int j = 0; j < THREAD_PER_BLOCK; j++)
        {
            cur += input[i * THREAD_PER_BLOCK + j];
        }
        res[i] = cur;
    }

    // 检查计算结果准确性
    check(out, res, BLOCK_NUM);
    return 0;
}
